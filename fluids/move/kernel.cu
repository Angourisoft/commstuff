#include "hip/hip_runtime.h"

#include "math_functions.h"
#include "cudart_platform.h"

#include <algorithm>
#include <iostream>
#include <cstdlib>

using uint8_t = unsigned char;

struct vec2
{
	float x = 0.0, y = 0.0;

	__device__ vec2 operator-(vec2 other)
	{
		vec2 res;
		res.x = this->x - other.x;
		res.y = this->y - other.y;
		return res;
	}

	__device__ vec2 operator+(vec2 other)
	{
		vec2 res;
		res.x = this->x + other.x;
		res.y = this->y + other.y;
		return res;
	}

	__device__ vec2 operator*(float d)
	{
		vec2 res;
		res.x = this->x * d;
		res.y = this->y * d;
		return res;
	}
};

struct Particle
{
	vec2 u; // velocity
	float q; // quantity
	float intensityR = 1.0f;
	float intensityG = 0.2f;
	float intensityB = 1.0f;
};

static Particle* cpuField;
static Particle* newField;
static Particle* oldField;
static uint8_t* colorField;
static size_t xSize, ySize;
static float* pressureOld;
static float* pressureNew;

// interpolates quantity of grid cells
__device__ vec2 interpolate(vec2 v, Particle* field, size_t xSize, size_t ySize)
{
	float x1 = (int)v.x;
	float y1 = (int)v.y;
	float x2 = (int)v.x + 1;
	float y2 = (int)v.y + 1;
	vec2 q1, q2, q3, q4;
	#define SET(Q, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) Q = field[int(y) * xSize + int(x)].u
	SET(q1, x1, y1);
	SET(q2, x1, y2);
	SET(q3, x2, y1);
	SET(q4, x2, y2);
	#undef SET
	vec2 f1 = q1 * ((x2 - v.x) / (x2 - x1)) + q3 * ((v.x - x1) / (x2 - x1));
	vec2 f2 = q2 * ((x2 - v.x) / (x2 - x1)) + q4 * ((v.x - x1) / (x2 - x1));
	return f1 * ((y2 - v.y) / (y2 - y1)) + f2 * ((v.y - y1) / (y2 - y1));
}

// performs iteration of jacobi method on grid field
__device__ vec2 jacobiVelocity(Particle* field, size_t xSize, size_t ySize, vec2 v, vec2 B, float alpha, float beta)
{
	vec2 vU, vD, vR, vL; 	     
	#define SET(U, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) U = field[int(y) * xSize + int(x)].u
	SET(vU, v.x, v.y - 1);
	SET(vD, v.x, v.y + 1);
	SET(vL, v.x - 1, v.y);
	SET(vR, v.x + 1, v.y);
	#undef SET
	v = (vU + vD + vL + vR + B * alpha) * (1.0f / beta);
	return v;
}

__device__ float jacobiPressure(float* pressureField, size_t xSize, size_t ySize, int x, int y, float B, float alpha, float beta)
{
	float xU = 0.0f, xD = 0.0f, xL = 0.0f, xR = 0.0f;
	#define SET(P, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) P = pressureField[int(y) * xSize + int(x)]
	SET(xU, x, y - 1);
	SET(xD, x, y + 1);
	SET(xL, x - 1, y);
	SET(xR, x + 1, y);
	#undef SET
	float pressure = (xU + xD + xL + xR + alpha * B) * (1.0f / beta);
	return pressure;
}

__device__ float divergency(Particle* field, size_t xSize, size_t ySize, int x, int y)
{
	float x1 = 0.0f, x2 = 0.0f, y1 = 0.0f, y2 = 0.0f;
	#define SET(P, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) P = field[int(y) * xSize + int(x)]
	SET(x1, x + 1, y).u.x;
	SET(x2, x - 1, y).u.x;
	SET(y1, x, y + 1).u.y;
	SET(y2, x, y - 1).u.y;
	#undef SET
	return (x1 - x2) / 2 + (y1 - y2) / 2;
}

__device__ vec2 gradient(float* pField, size_t xSize, size_t ySize, int x, int y)
{
	#define SET(P, x, y) if (x < xSize && x >= 0 && y < ySize && y >= 0) P = pField[int(y) * xSize + int(x)]
	float x1 = 0.0f, x2 = 0.0f, y1 = 0.0f, y2 = 0.0f;
	SET(x1, x + 1, y);
	SET(x2, x - 1, y);
	SET(y1, x, y + 1);
	SET(y2, x, y - 1);
	#undef SET
	vec2 res = { (x1 - x2) / 2.0f, (y1 - y2) / 2.0f };
	return res;	 
}

__device__ float sigm(float x)
{
	return 1.0f / (1.0f + powf(1.2f, -x));
}

// adds quantity to particles using bilinear interpolation
__global__ void advect(Particle* newField, Particle* oldField, size_t xSize, size_t ySize, float dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	vec2 pos = { x * 1.0f, y * 1.0f };
	Particle& Pnew = newField[y * xSize + x];
	Particle& Pold = oldField[y * xSize + x];
	// find new quantity tracing where it came from
	Pnew.u = interpolate(pos - Pold.u * dt, oldField, xSize, ySize);
}

__global__ void paint(uint8_t* colorField, Particle* field, size_t xSize, size_t ySize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float q = (sigm(sqrt(powf(field[y * xSize + x].u.x, 2) + powf(field[y * xSize + x].u.y, 2))) - 0.5) * 2;
	float R = field[y * xSize + x].intensityR;
	float G = field[y * xSize + x].intensityG;
	float B = field[y * xSize + x].intensityB;
	colorField[4 * (y * xSize + x) + 0] = 255 * powf(q, 4.0f);
	colorField[4 * (y * xSize + x) + 1] = 255 * powf(q, 0.4f);
	colorField[4 * (y * xSize + x) + 2] = 255 * powf(q, 4.0f);
	colorField[4 * (y * xSize + x) + 3] = 255;
}

// calculates nonzero divergency velocity field u
__global__ void diffuse(Particle* newField, Particle* oldField, size_t xSize, size_t ySize, float viscosity, float dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	vec2 pos = { x * 1.0f, y * 1.0f };
	vec2 u = oldField[y * xSize + x].u;
	// perfom one iteration of jacobi method (diffuse method should be called 20-50 times per cell)
	float alpha = viscosity * viscosity / dt;
	float beta = 4.0f + alpha;
	newField[y * xSize + x].u = jacobiVelocity(oldField, xSize, ySize, pos, u, alpha, beta);
}

__global__ void computePressure(Particle* newField, size_t xSize, size_t ySize, float* pNew, float* pOld, float density, float dt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float div = divergency(newField, xSize, ySize, x, y);
	float alpha = -1.0f * density * density;
	float beta = 4.0;
	pNew[y * xSize + x] = jacobiPressure(pOld, xSize, ySize, x, y, div, alpha, beta);
}

__global__ void project(Particle* newField, size_t xSize, size_t ySize, float* pField)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	vec2& u = newField[y * xSize + x].u;
	u = u - gradient(pField, xSize, ySize, x, y);
}

float randomf()
{
	return rand() * 1.0f / RAND_MAX;
}

void applyForce(int x1, int y1, int x2, int y2, int r, float R, float G, float B)
{
	hipMemcpy(cpuField, oldField, xSize * ySize * sizeof(Particle), hipMemcpyDeviceToHost);
	float color = randomf();
	float length = std::sqrtf(powf(x2 - x1, 2) + powf(y2 - y1, 2)) + 1;
	for (int dx = -r; dx < r; dx++)
	{
		for (int dy = -r; dy < r; dy++)
		{
			if (dx * dx + dy * dy < r * r)
			{
				int ax = std::max(0, std::min(int(xSize) - 1, x1 + dx));
				int ay = std::max(0, std::min(int(ySize) - 1, y1 + dy));
				vec2& u = cpuField[ay * xSize + ax].u;
				u.x += (x2 - x1) * 100 / length;
				//u.x = randomf() * 10;
				u.y += (y2 - y1) * 100 / length;
				
				//u.x = randomf() * 10.0f;
				//u.y = randomf() * 10.0f;
				cpuField[ay * xSize + ax].q = color;
				/*
				cpuField[ay * xSize + ax].intensityR = R;
				cpuField[ay * xSize + ax].intensityG = G;
				cpuField[ay * xSize + ax].intensityB = B;
				*/
			}
		}
	}
	hipMemcpy(oldField, cpuField, xSize * ySize * sizeof(Particle), hipMemcpyHostToDevice);
}

void cudaInit(size_t x, size_t y)
{
	xSize = x, ySize = y;
	hipSetDevice(0);
	size_t size = xSize * ySize * 4 * sizeof(uint8_t);
	hipMalloc(&colorField, size);
	hipMalloc(&oldField, xSize * ySize * sizeof(Particle));
	hipMalloc(&newField, xSize * ySize * sizeof(Particle));
	hipMalloc(&pressureOld, xSize * ySize * sizeof(float));
	hipMalloc(&pressureNew, xSize * ySize * sizeof(float));

	hipMemset(oldField, 0, xSize * ySize * sizeof(Particle));
	hipMemset(pressureOld, 0, xSize * ySize * sizeof(float));

	cpuField = new Particle[xSize * ySize];
}

void cudaExit()
{
	delete[] cpuField;
	hipFree(colorField);
	hipFree(oldField);
	hipFree(newField);
	hipFree(pressureOld);
	hipFree(pressureNew);
	hipDeviceReset();
}

void computeField(uint8_t* result, float dt, float viscosity, float density)
{
	int iterations = 50;
	dim3 threadsPerBlock(20, 20);
	dim3 numBlocks(xSize / threadsPerBlock.x, ySize / threadsPerBlock.y);
	// run advect -> diffuse -> force -> project
	advect<<<numBlocks, threadsPerBlock>>>(newField, oldField, xSize, ySize, dt);
	hipDeviceSynchronize();
	std::swap(newField, oldField);

#if 0
	Particle* tmp = new Particle[xSize * ySize];
	cudaMemcpy(tmp, newField, xSize * ySize * sizeof(Particle), cudaMemcpyDeviceToHost);
	printf("%f ", tmp[100 * ySize + 100].q);
	std::cout << '[' << tmp[100 * ySize + 100].u.x << ' ';
	std::cout		 << tmp[100 * ySize + 100].u.y << "]\n";
	delete[] tmp;
#endif
	
	paint<<<numBlocks, threadsPerBlock>>>(colorField, newField, xSize, ySize);
	hipDeviceSynchronize();
	for (int i = 0; i < iterations; i++)
	{
		diffuse<<<numBlocks, threadsPerBlock>>>(newField, oldField, xSize, ySize, viscosity, dt);
		hipDeviceSynchronize();
		std::swap(newField, oldField);
	}
	for (int i = 0; i < iterations; i++)
	{
		computePressure<<<numBlocks, threadsPerBlock>>>(newField, xSize, ySize, pressureNew, pressureOld, density, dt);
		hipDeviceSynchronize();
		std::swap(pressureNew, pressureOld);
	}
	project<<<numBlocks, threadsPerBlock>>>(newField, xSize, ySize, pressureOld);
	hipDeviceSynchronize();
	hipMemset(pressureOld, 0, xSize * ySize * sizeof(float));

	size_t size = xSize * ySize * 4 * sizeof(uint8_t);
	hipMemcpy(result, colorField, size, hipMemcpyDeviceToHost);
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		std::cout << hipGetErrorName(error) << std::endl;
	}
}